#include <iostream>
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

int main() {
    #if  __CUDA_ARCH__ < 600 and defined(__CUDA_ARCH__)
        printf("CUDA_ARCH: %d", __CUDA_ARCH__)
    #endif
}